#include "hip/hip_runtime.h"
#include "gpumat_find_nonzero.cuh"
#include <cmath>



__global__  
void kernelFindNonzero(unsigned char* gpu_image_data, int rows, int cols, int step,
	PointXY* nonzero_xy_coords, int* gpu_num_nonzeros)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y; //Row number
	int j = blockIdx.x * blockDim.x + threadIdx.x; //Column number

	int cur_ind = 0;
	//�̹��� ���� ���� �ε��� --> 0�� �ƴ� ��� nonzero�� ����
	//atomicAdd�� ���� �ߺ� �����̳� �޸� ���� ������ ����
	if (i < rows && j < cols)
	{
		if (gpu_image_data[i * step + j] != unsigned char(0))
		{
			cur_ind = atomicAdd(&gpu_num_nonzeros[0], 1);
			nonzero_xy_coords[cur_ind].x = j;
			nonzero_xy_coords[cur_ind].y = i;
		}

	}
}

GpuMatFindNonzero::GpuMatFindNonzero(int rows, int cols, int step, int max_num_nonzeros, hipStream_t* stream)
{
	rows_ = rows;
	cols_ = cols;
	step_ = step;
	max_num_nozneros_ = max_num_nonzeros;

	//consider warp thread size
	row_block_size_ = int(rows/32+1);
	col_block_size_ = int(cols/32+1);

	grid_ = dim3(col_block_size_, row_block_size_);
	block_  = dim3(32, 32);

	cpu_num_nonzeros_ = new int[1];
	cpu_num_nonzeros_[0] = 0;
	cpu_nonzero_xy_coords_ = nullptr;

	stream_ = stream;

	HANDLE_ERROR(hipMallocAsync((void**)&gpu_num_nonzeros_, sizeof(int), stream_[0]));
	HANDLE_ERROR(hipMemsetAsync(gpu_num_nonzeros_, 0, sizeof(int), stream_[0]));
	HANDLE_ERROR(hipMallocAsync((void**)&gpu_nonzero_xy_coords_, 2 * max_num_nozneros_ * sizeof(int), stream_[0]));

	HANDLE_ERROR(hipStreamSynchronize(stream_[0]));
}

GpuMatFindNonzero::~GpuMatFindNonzero()
{
	HANDLE_ERROR(hipFreeAsync(gpu_num_nonzeros_, stream_[0]));
	HANDLE_ERROR(hipFreeAsync(gpu_nonzero_xy_coords_, stream_[0]));

	delete[] cpu_num_nonzeros_;

	if (cpu_nonzero_xy_coords_ != nullptr)
		delete[] cpu_nonzero_xy_coords_;

	HANDLE_ERROR(hipStreamSynchronize(stream_[0]));
}

void GpuMatFindNonzero::findNonzero(cv::cuda::GpuMat gpu_image, bool is_validation)
{	
	HANDLE_ERROR(hipMemsetAsync(gpu_num_nonzeros_, 0, sizeof(int), stream_[0]));

	kernelFindNonzero <<<grid_, block_, 0, stream_[0]>>> (reinterpret_cast<unsigned char*>(gpu_image.data), rows_, 
														 cols_, step_, gpu_nonzero_xy_coords_, gpu_num_nonzeros_);

	HANDLE_ERROR(hipMemcpyAsync(&cpu_num_nonzeros_[0], &gpu_num_nonzeros_[0], sizeof(int), 
						      hipMemcpyDeviceToHost, stream_[0]));
	
	if (is_validation)
	{
		HANDLE_ERROR(hipStreamSynchronize(stream_[0]));
		if (cpu_nonzero_xy_coords_ != nullptr)
			delete[] cpu_nonzero_xy_coords_;
		cpu_nonzero_xy_coords_ = new PointXY[cpu_num_nonzeros_[0]];
		HANDLE_ERROR(hipMemcpyAsync(&cpu_nonzero_xy_coords_[0], &gpu_nonzero_xy_coords_[0], cpu_num_nonzeros_[0] * sizeof(PointXY), hipMemcpyDeviceToHost, stream_[0]));
	}

	HANDLE_ERROR(hipStreamSynchronize(stream_[0]));

	num_nonzeros_ = cpu_num_nonzeros_[0];
	return;
}