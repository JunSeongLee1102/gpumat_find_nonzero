#include "hip/hip_runtime.h"
#include "gpumat_find_nonzero.cuh"
#include <cmath>



__global__  
void kernelFindNonzero(unsigned char* gpu_image_data, int rows, int cols, int step,
	PointXY* nonzero_xy_coords, int* gpu_num_nonzeros)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y; //Row number
	int j = blockIdx.x * blockDim.x + threadIdx.x; //Column number

	int cur_ind = 0;
	//�̹��� ���� ���� �ε��� --> 0�� �ƴ� ��� nonzero�� ����
	//atomicAdd�� ���� �ߺ� �����̳� �޸� ���� ������ ����
	if (i < rows && j < cols)
	{
		if (gpu_image_data[i * step + j] != unsigned char(0))
		{
			cur_ind = atomicAdd(&gpu_num_nonzeros[0], 1);
			nonzero_xy_coords[cur_ind].x = j;
			nonzero_xy_coords[cur_ind].y = i;
		}

	}
}

GpuMatFindNonzero::GpuMatFindNonzero(int rows, int cols, int step, int max_num_nonzeros, hipStream_t* stream)
{
	_rows = rows;
	_cols = cols;
	_step = step;
	_max_num_nonzeros = max_num_nonzeros;

	//consider warp thread size
	_row_block_size = int(rows/32+1);
	_col_block_size = int(cols/32+1);

	_grid = dim3(_col_block_size, _row_block_size);
	_block  = dim3(32, 32);

	_cpu_num_nonzeros = new int[1];
	_cpu_num_nonzeros[0] = 0;
	_cpu_nonzero_xy_coords = nullptr;

	_stream = stream;

	HANDLE_ERROR(hipMallocAsync((void**)&_gpu_num_nonzeros, sizeof(int), _stream[0]));
	HANDLE_ERROR(hipMemsetAsync(_gpu_num_nonzeros, 0, sizeof(int), _stream[0]));
	HANDLE_ERROR(hipMallocAsync((void**)&_gpu_nonzero_xy_coords, 2 * _max_num_nonzeros * sizeof(int), _stream[0]));

	HANDLE_ERROR(hipStreamSynchronize(_stream[0]));
}

GpuMatFindNonzero::~GpuMatFindNonzero()
{
	HANDLE_ERROR(hipFreeAsync(_gpu_num_nonzeros, _stream[0]));
	HANDLE_ERROR(hipFreeAsync(_gpu_nonzero_xy_coords, _stream[0]));

	delete[] _cpu_num_nonzeros;

	if (_cpu_nonzero_xy_coords != nullptr)
		delete[] _cpu_nonzero_xy_coords;

	HANDLE_ERROR(hipStreamSynchronize(_stream[0]));
}

void GpuMatFindNonzero::findNonzero(cv::cuda::GpuMat gpu_image, bool is_validation)
{	
	HANDLE_ERROR(hipMemsetAsync(_gpu_num_nonzeros, 0, sizeof(int), _stream[0]));

	kernelFindNonzero <<<_grid, _block, 0, _stream[0]>>> (reinterpret_cast<unsigned char*>(gpu_image.data), _rows, 
														 _cols, _step, _gpu_nonzero_xy_coords, _gpu_num_nonzeros);

	HANDLE_ERROR(hipMemcpyAsync(&_cpu_num_nonzeros[0], &_gpu_num_nonzeros[0], sizeof(int), 
						      hipMemcpyDeviceToHost, _stream[0]));
	
	if (is_validation)
	{
		HANDLE_ERROR(hipStreamSynchronize(_stream[0]));
		if (_cpu_nonzero_xy_coords != nullptr)
			delete[] _cpu_nonzero_xy_coords;
		_cpu_nonzero_xy_coords = new PointXY[_cpu_num_nonzeros[0]];
		HANDLE_ERROR(hipMemcpyAsync(&_cpu_nonzero_xy_coords[0], &_gpu_nonzero_xy_coords[0], _cpu_num_nonzeros[0] * sizeof(PointXY), hipMemcpyDeviceToHost, _stream[0]));
	}

	HANDLE_ERROR(hipStreamSynchronize(_stream[0]));

	_num_nonzeros = _cpu_num_nonzeros[0];
	return;
}